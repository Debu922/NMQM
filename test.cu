
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>();
    printf("Hello World from GPU!\n");

    return 0;
}

